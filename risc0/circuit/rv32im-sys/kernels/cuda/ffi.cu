#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "context.h"
#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "kernels.h"

#include "vendor/nvtx3/nvtx3.hpp"

#include <cstring>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

constexpr size_t kVerifyMemBodyKind = 1;
constexpr size_t kVerifyMemHaltKind = 2;

LaunchConfig getSimpleConfig(uint32_t count) {
  int device;
  CUDA_OK(hipGetDevice(&device));

  int maxThreads;
  CUDA_OK(hipDeviceGetAttribute(&maxThreads, hipDeviceAttributeMaxThreadsPerBlock, device));

  int block = maxThreads / 4;
  int grid = (count + block - 1) / block;
  return LaunchConfig{grid, block, 0};
}

struct Error {
  const char* msg;
};

struct HostContext {
  MachineContext* ctx;

  HostContext(PreflightTrace* trace, size_t steps) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(MachineContext)));
    ctx->steps = steps;

    CUDA_OK(hipMallocManaged(&ctx->trace, sizeof(PreflightTrace)));
    ctx->trace->isTrace = trace->isTrace;

    // ctx->trace->numCycles = trace->numCycles;
    // printf("numCycles: %u\n", trace->numCycles);
    CUDA_OK(hipMalloc(&ctx->trace->cycles, trace->numCycles * sizeof(PreflightCycle)));
    CUDA_OK(hipMemcpy(ctx->trace->cycles,
                       trace->cycles,
                       trace->numCycles * sizeof(PreflightCycle),
                       hipMemcpyHostToDevice));

    // ctx->trace->numTxns = trace->numTxns;
    // printf("numTxns: %u\n", trace->numTxns);
    CUDA_OK(hipMalloc(&ctx->trace->txns, trace->numTxns * sizeof(MemoryTransaction)));
    CUDA_OK(hipMemcpy(ctx->trace->txns,
                       trace->txns,
                       trace->numTxns * sizeof(MemoryTransaction),
                       hipMemcpyHostToDevice));

    // ctx->trace->numExtras = trace->numExtras;
    // printf("numExtras: %u\n", trace->numExtras);
    CUDA_OK(hipMalloc(&ctx->trace->extras, trace->numExtras * sizeof(uint32_t)));
    CUDA_OK(hipMemcpy(ctx->trace->extras,
                       trace->extras,
                       trace->numExtras * sizeof(uint32_t),
                       hipMemcpyHostToDevice));

    ctx->h_ramRows.resize(steps * kMaxRamRowsPerCycle);
    CUDA_OK(hipMalloc(&ctx->ramRows, steps * kMaxRamRowsPerCycle * sizeof(RamArgumentRow)));
    CUDA_OK(hipMemset(
        ctx->ramRows, kInvalidPattern, steps * kMaxRamRowsPerCycle * sizeof(RamArgumentRow)));
    CUDA_OK(hipMalloc(&ctx->ramIndex, steps * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->ramIndex, 0, steps * sizeof(uint32_t)));
    CUDA_OK(hipMalloc(&ctx->pairs, steps * kMaxBytePairsPerCycle * sizeof(uint32_t)));
    CUDA_OK(
        hipMemset(ctx->pairs, kInvalidPattern, steps * kMaxBytePairsPerCycle * sizeof(uint32_t)));
    CUDA_OK(hipMalloc(&ctx->pairsIndex, steps * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->pairsIndex, 0, steps * sizeof(uint32_t)));
  }

  ~HostContext() {
    // printf("~HostContext\n");
    hipFree(ctx->trace->cycles);
    hipFree(ctx->trace->txns);
    hipFree(ctx->trace->extras);
    hipFree(ctx->trace);
    hipFree(ctx->ramRows);
    hipFree(ctx->ramIndex);
    hipFree(ctx->pairs);
    hipFree(ctx->pairsIndex);
    hipFree(ctx);
  }
};

__device__ bool MachineContext::isParSafeExec(uint32_t cycle) const {
  return trace->cycles[cycle].isSafeExec;
}

__device__ uint8_t MachineContext::isParSafeVerifyMem(uint32_t cycle) const {
  return trace->cycles[cycle].isSafeVerifyMem;
}

__global__ void
par_step_exec(MachineContext* ctx, uint32_t steps, uint32_t count, Fp* arg0, Fp* arg1, Fp* arg2) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == 0 || ctx->isParSafeExec(cycle)) {
    // printf("step_exec(%u)\n", cycle);
    step_exec(ctx, steps, cycle++, arg0, arg1, arg2, nullptr, nullptr);
    while (cycle < count && !ctx->isParSafeExec(cycle)) {
      // printf("step_exec(%u)\n", cycle);
      step_exec(ctx, steps, cycle++, arg0, arg1, arg2, nullptr, nullptr);
    }
  }
}

__global__ void par_step_verify_mem(
    MachineContext* ctx, uint32_t steps, uint32_t count, Fp* ctrl, Fp* io, Fp* data) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == 0 || ctx->isParSafeVerifyMem(cycle)) {
    // printf("step_verify_mem(%u)\n", cycle);
    step_verify_mem(ctx, steps, cycle++, ctrl, io, data, nullptr, nullptr);
    while (cycle < count && !ctx->isParSafeVerifyMem(cycle)) {
      step_verify_mem(ctx, steps, cycle++, ctrl, io, data, nullptr, nullptr);
    }
  }
}

void MachineContext::sortRam() {
  // printf("sortRam\n");
  nvtx3::scoped_range range("sortRam");
  {
    nvtx3::scoped_range range("sort");
    thrust::sort(thrust::device, ramRows, ramRows + steps * kMaxRamRowsPerCycle);
  }

  {
    nvtx3::scoped_range range("dirty");
    CUDA_OK(hipMemcpy(h_ramRows.data(),
                       ramRows,
                       h_ramRows.size() * sizeof(RamArgumentRow),
                       hipMemcpyDeviceToHost));

    uint32_t prevDirty = 0;
    for (size_t i = 0; i < steps * kMaxRamRowsPerCycle; i++) {
      RamArgumentRow& row = h_ramRows[i];
      switch (row.getMemOp()) {
      case 0: // pageIo
        row.dirty = 0;
        break;
      case 1: // read
        row.dirty = prevDirty;
        break;
      case 2: // write
        row.dirty = 1;
        break;
      }
      prevDirty = row.dirty;
    }

    CUDA_OK(hipMemcpy(ramRows,
                       h_ramRows.data(),
                       h_ramRows.size() * sizeof(RamArgumentRow),
                       hipMemcpyHostToDevice));
  }

  {
    nvtx3::scoped_range range("scan");
    thrust::exclusive_scan(thrust::device, ramIndex, ramIndex + steps, ramIndex, 0);
  }
}

__global__ void inject_backs_ram(MachineContext* ctx, uint32_t steps, uint32_t count, Fp* data) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  uint8_t kind = ctx->isParSafeVerifyMem(cycle);
  if (cycle > 1 && kind) {
    size_t idx = ctx->ramIndex[cycle];
    assert(idx != 0);

    const RamArgumentRow& back1 = ctx->ramRows[idx - 1];
    data[89 * steps + cycle - 1] = back1.addr;              // a->addr
    data[90 * steps + cycle - 1] = back1.getMemCycle();     // a->cycle
    data[91 * steps + cycle - 1] = back1.getMemOp();        // a->memOp
    data[92 * steps + cycle - 1] = back1.word & 0xff;       // a->data[0]
    data[93 * steps + cycle - 1] = back1.word >> 8 & 0xff;  // a->data[1]
    data[94 * steps + cycle - 1] = back1.word >> 16 & 0xff; // a->data[2]
    data[95 * steps + cycle - 1] = back1.word >> 24 & 0xff; // a->data[3]
    data[97 * steps + cycle - 1] = back1.dirty;             // prevVerifier->dirty
    if (kind == kVerifyMemHaltKind) {
      const RamArgumentRow& back2 = ctx->ramRows[idx - 2];
      uint32_t isNewAddr = back2.addr != back1.addr;
      uint32_t cmp;
      if (isNewAddr) {
        cmp = back1.addr - back2.addr - 1;
      } else {
        cmp =
            back1.getMemCycle() * 3 + back1.getMemOp() - back2.getMemCycle() * 3 + back2.getMemOp();
      }
      uint32_t diff[3];
      for (size_t i = 0; i < 3; i++) {
        diff[i] = cmp & 0xff;
        cmp = cmp >> 8;
      }
      uint32_t extra = cmp;
      data[96 * steps + cycle - 1] = isNewAddr; // isNewAddr
      data[3 * steps + cycle - 1] = diff[0];    // diff[0]
      data[4 * steps + cycle - 1] = diff[1];    // diff[1]
      data[5 * steps + cycle - 1] = diff[2];    // diff[2]
      data[69 * steps + cycle - 1] = extra;     // extra
    }
  }
}

void MachineContext::sortBytes() {
  nvtx3::scoped_range range("sortBytes");

  {
    nvtx3::scoped_range range("sort");
    thrust::sort(thrust::device, pairs, pairs + steps * kMaxBytePairsPerCycle);
  }

  {
    nvtx3::scoped_range range("scan");
    thrust::exclusive_scan(thrust::device, pairsIndex, pairsIndex + steps, pairsIndex, 0);
  }
}

__global__ void inject_backs_bytes(void* ctx, size_t steps, size_t count, Fp* data) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle == 0 || cycle >= count) {
    return;
  }

  MachineContext* mctx = static_cast<MachineContext*>(ctx);
  uint32_t idx = mctx->pairsIndex[cycle];
  uint32_t writeCount = idx - mctx->pairsIndex[cycle - 1];
  // printf("inject> cycle: %u, writeCount: %lu\n", cycle, writeCount);
  if (writeCount) {
    uint32_t pair = mctx->pairs[idx - 1];
    // printf("inject> pair: %x\n", pair);
    data[0 * steps + cycle - 1] = pair >> 8 & 0xff;
    data[1 * steps + cycle - 1] = pair & 0xff;
  }
}

extern "C" {

const char* risc0_circuit_rv32im_cuda_witgen(
    PreflightTrace* trace, uint32_t steps, uint32_t last_cycle, Fp* ctrl, Fp* io, Fp* data) {
  try {
    nvtx3::scoped_range range("witgen");

    // printf("risc0_circuit_rv32im_cuda_witgen\n");
    CUDA_OK(hipDeviceSynchronize());

    HostContext ctx(trace, steps);

    CudaStream stream;
    LaunchConfig cfg = getSimpleConfig(last_cycle);

    {
      // printf("step_exec\n");
      nvtx3::scoped_range range("step_exec");
      par_step_exec<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, ctrl, io, data);
      CUDA_OK(hipStreamSynchronize(stream));
    }

    {
      nvtx3::scoped_range range("verify_ram");
      ctx.ctx->sortRam();

      {
        // printf("inject_backs_ram\n");
        nvtx3::scoped_range range("inject_backs_ram");
        inject_backs_ram<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, data);
        CUDA_OK(hipStreamSynchronize(stream));
      }

      {
        // printf("step_verify_mem\n");
        nvtx3::scoped_range range("step_verify_mem");
        par_step_verify_mem<<<cfg.grid, cfg.block, 0, stream>>>(
            ctx.ctx, steps, last_cycle, ctrl, io, data);
        CUDA_OK(hipStreamSynchronize(stream));
      }
    }

    {
      nvtx3::scoped_range range("verify_bytes");
      ctx.ctx->sortBytes();

      {
        // printf("inject_backs_bytes\n");
        nvtx3::scoped_range range("inject_backs_bytes");
        inject_backs_bytes<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, data);
        CUDA_OK(hipStreamSynchronize(stream));
      }

      {
        // printf("step_verify_bytes\n");
        nvtx3::scoped_range range("step_verify_bytes");
        step_verify_bytes<<<cfg.grid, cfg.block, 0, stream>>>(
            ctx.ctx, steps, last_cycle, ctrl, io, data, nullptr, nullptr);
        CUDA_OK(hipStreamSynchronize(stream));
      }
    }

  } catch (const std::exception& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_step_verify_bytes(
    void* ctx, uint32_t steps, uint32_t count, Fp* ctrl, Fp* io, Fp* data) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    step_verify_bytes<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, nullptr, nullptr);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_step_compute_accum(
    void* ctx, uint32_t steps, uint32_t count, Fp* ctrl, Fp* io, Fp* data, Fp* mix, Fp* accum) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    step_compute_accum<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, mix, accum);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_step_verify_accum(
    void* ctx, uint32_t steps, uint32_t count, Fp* ctrl, Fp* io, Fp* data, Fp* mix, Fp* accum) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    step_verify_accum<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, mix, accum);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_eval_check(Fp* check,
                                                 const Fp* ctrl,
                                                 const Fp* data,
                                                 const Fp* accum,
                                                 const Fp* mix,
                                                 const Fp* out,
                                                 const Fp& rou,
                                                 uint32_t po2,
                                                 uint32_t domain,
                                                 const FpExt* poly_mix_pows) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

} // extern "C"
