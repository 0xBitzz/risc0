// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "fp.h"
#include "fpext.h"
#include "kernels.h"

#include <cstring>
#include <hip/hip_runtime.h>
#include <stdexcept>

LaunchConfig getSimpleConfig(uint32_t count) {
  int device;
  CUDA_OK(hipGetDevice(&device));

  int maxThreads;
  CUDA_OK(hipDeviceGetAttribute(&maxThreads, hipDeviceAttributeMaxThreadsPerBlock, device));

  int block = maxThreads / 4;
  int grid = (count + block - 1) / block;
  return LaunchConfig{grid, block, 0};
}

extern "C" {

const char* risc0_circuit_rv32im_cuda_step_verify_bytes(
    void* ctx, uint32_t steps, uint32_t count, Fp* ctrl, Fp* io, Fp* data) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    step_verify_bytes<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, nullptr, nullptr);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_step_compute_accum(
    void* ctx, uint32_t steps, uint32_t count, Fp* ctrl, Fp* io, Fp* data, Fp* mix, Fp* accum) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    step_compute_accum<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, mix, accum);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_step_verify_accum(
    void* ctx, uint32_t steps, uint32_t count, Fp* ctrl, Fp* io, Fp* data, Fp* mix, Fp* accum) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(count);
    step_verify_accum<<<cfg.grid, cfg.block, 0, stream>>>(
        ctx, steps, count, ctrl, io, data, mix, accum);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

const char* risc0_circuit_rv32im_cuda_eval_check(Fp* check,
                                                 const Fp* ctrl,
                                                 const Fp* data,
                                                 const Fp* accum,
                                                 const Fp* mix,
                                                 const Fp* out,
                                                 const Fp& rou,
                                                 uint32_t po2,
                                                 uint32_t domain,
                                                 const FpExt* poly_mix_pows) {
  try {
    CUDA_OK(hipDeviceSynchronize());

    CudaStream stream;
    auto cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::runtime_error& err) {
    return strdup(err.what());
  }
  return nullptr;
}

} // extern "C"
