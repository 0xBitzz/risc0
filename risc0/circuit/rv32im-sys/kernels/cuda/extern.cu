#include "hip/hip_runtime.h"
// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#if defined(__clang__)
#pragma clang diagnostic ignored "-Wunused-function"
#pragma clang diagnostic ignored "-Wunused-parameter"
#pragma clang diagnostic ignored "-Wunused-variable"
#elif defined(__GNUC__)
#pragma GCC diagnostic ignored "-Wunused-function"
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wunused-variable"
#endif

#include "hip/hip_runtime.h"
#include "extern.h"
#include "fpext.h"
#include "kernels.h"

#include "vendor/nvtx3/nvtx3.hpp"

#include <assert.h>
#include <cstdint>
#include <cuda/std/array>
#include <cuda/std/atomic>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <vector>

constexpr size_t kBabyBearExtSize = 4;
constexpr size_t kMaxRamRowsPerCycle = 5;
constexpr size_t kMaxBytePairsPerCycle = 21;
constexpr size_t kTotalBytePairs = 256 * 256;

constexpr size_t kWordSize = sizeof(uint32_t);
constexpr size_t kBitWidth = 256;
constexpr size_t kByteWidth = kBitWidth / 8;

struct MemoryTransaction {
  uint32_t cycle;
  uint32_t addr;
  uint32_t data;
};

struct PreflightCycle {
  uint8_t major;
  uint8_t minor;
  uint8_t isSafeExec;
  uint8_t isSafeVerifyMem;
  uint32_t memIdx;
  uint32_t extraIdx;
};

struct PreflightTrace {
  PreflightCycle* cycles;
  MemoryTransaction* txns;
  uint32_t* extras;
  uint32_t numCycles;
  uint32_t numTxns;
  uint32_t numExtras;
  uint32_t isTrace;
};

struct RamArgumentRow {
  uint32_t addr;
  uint32_t cyclop;
  uint32_t word;
  uint32_t dirty;

  __device__ void setCyclop(uint32_t memCycle, uint32_t memOp) { cyclop = (memCycle << 2) | memOp; }
  __device__ uint32_t getMemCycle() const { return cyclop >> 2; }
  __host__ __device__ uint32_t getMemOp() const { return cyclop & 0b11; }

  __device__ bool operator<(const RamArgumentRow& other) const {
    thrust::tuple<uint32_t, uint32_t, uint32_t, uint32_t> lhs(addr, cyclop, word, dirty);
    thrust::tuple<uint32_t, uint32_t, uint32_t, uint32_t> rhs(
        other.addr, other.cyclop, other.word, other.dirty);
    return lhs < rhs;
  }
};

struct MachineContext {
  PreflightTrace* trace;
  size_t steps;

  RamArgumentRow* ramRows;
  RamArgumentRow* ramSorted;
  uint32_t* ramIndex;

  cuda::std::atomic<uint32_t>* bytePairs;
  uint32_t* byteSorted;
  uint32_t* byteWrites;
  uint32_t* byteReads;

  void sortRam();
  void sortBytes();
};

struct AccumContext {
  FpExt* ram;
  FpExt* bytes;
};

struct Error {
  const char* msg;
};

struct HostContext {
  MachineContext* ctx;

  HostContext(PreflightTrace* trace, size_t steps) {
    CUDA_OK(hipMallocManaged(&ctx, sizeof(MachineContext)));
    ctx->steps = steps;

    CUDA_OK(hipMallocManaged(&ctx->trace, sizeof(PreflightTrace)));
    ctx->trace->isTrace = trace->isTrace;

    // ctx->trace->numCycles = trace->numCycles;
    // printf("numCycles: %u\n", trace->numCycles);
    CUDA_OK(hipMalloc(&ctx->trace->cycles, trace->numCycles * sizeof(PreflightCycle)));
    CUDA_OK(hipMemcpy(ctx->trace->cycles,
                       trace->cycles,
                       trace->numCycles * sizeof(PreflightCycle),
                       hipMemcpyHostToDevice));

    // ctx->trace->numTxns = trace->numTxns;
    // printf("numTxns: %u\n", trace->numTxns);
    CUDA_OK(hipMalloc(&ctx->trace->txns, trace->numTxns * sizeof(MemoryTransaction)));
    CUDA_OK(hipMemcpy(ctx->trace->txns,
                       trace->txns,
                       trace->numTxns * sizeof(MemoryTransaction),
                       hipMemcpyHostToDevice));

    // ctx->trace->numExtras = trace->numExtras;
    // printf("numExtras: %u\n", trace->numExtras);
    CUDA_OK(hipMalloc(&ctx->trace->extras, trace->numExtras * sizeof(uint32_t)));
    CUDA_OK(hipMemcpy(ctx->trace->extras,
                       trace->extras,
                       trace->numExtras * sizeof(uint32_t),
                       hipMemcpyHostToDevice));

    CUDA_OK(hipMallocManaged(&ctx->ramRows, steps * kMaxRamRowsPerCycle * sizeof(RamArgumentRow)));
    ctx->ramSorted = nullptr; // allocated later in sortRam
    CUDA_OK(hipMallocManaged(&ctx->ramIndex, steps * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->ramIndex, 0, steps * sizeof(uint32_t)));
    CUDA_OK(hipMallocManaged(&ctx->bytePairs, kTotalBytePairs * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->bytePairs, 0, kTotalBytePairs * sizeof(uint32_t)));
    CUDA_OK(hipMallocManaged(&ctx->byteSorted, steps * kMaxBytePairsPerCycle * sizeof(uint32_t)));
    CUDA_OK(hipMallocManaged(&ctx->byteWrites, steps * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->byteWrites, 0, steps * sizeof(uint32_t)));
    CUDA_OK(hipMalloc(&ctx->byteReads, steps * sizeof(uint32_t)));
    CUDA_OK(hipMemset(ctx->byteReads, 0, steps * sizeof(uint32_t)));
  }

  ~HostContext() {
    // printf("~HostContext\n");
    hipFree(ctx->trace->cycles);
    hipFree(ctx->trace->txns);
    hipFree(ctx->trace->extras);
    hipFree(ctx->trace);
    hipFree(ctx->ramRows);
    hipFree(ctx->ramSorted);
    hipFree(ctx->ramIndex);
    hipFree(ctx->byteSorted);
    hipFree(ctx->byteReads);
    hipFree(ctx);
  }
};

template <typename T, typename F> T ffi_wrap(Error* err, T val, F fn) {
  try {
    err->msg = nullptr;
    return fn();
  } catch (const std::exception& ex) {
    err->msg = strdup(ex.what());
    return val;
  }
}

__device__ bool is_par_safe_exec(void* ctx, uint32_t cycle) {
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  return trace->cycles[cycle].isSafeExec;
}

__device__ bool is_par_safe_verify_mem(void* ctx, uint32_t cycle) {
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  return trace->cycles[cycle].isSafeVerifyMem;
}

__global__ void
par_step_exec(void* ctx, uint32_t steps, uint32_t count, Fp* arg0, Fp* arg1, Fp* arg2) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == 0 || is_par_safe_exec(ctx, cycle)) {
    // printf("step_exec(%u)\n", cycle);
    step_exec(ctx, steps, cycle++, arg0, arg1, arg2, nullptr, nullptr);
    while (cycle < count && !is_par_safe_exec(ctx, cycle)) {
      // printf("step_exec(%u)\n", cycle);
      step_exec(ctx, steps, cycle++, arg0, arg1, arg2, nullptr, nullptr);
    }
  }
}

__global__ void inject_backs_ram(void* ctx, uint32_t steps, uint32_t count, Fp* data) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle > 0 && is_par_safe_verify_mem(ctx, cycle)) {
    MachineContext* mctx = static_cast<MachineContext*>(ctx);
    size_t idx = mctx->ramIndex[cycle];
    assert(idx != 0);

    const RamArgumentRow& row = mctx->ramSorted[idx - 1];
    data[89 * steps + cycle - 1] = row.addr;
    data[90 * steps + cycle - 1] = row.getMemCycle();     // a->cycle
    data[91 * steps + cycle - 1] = row.getMemOp();        // a->memOp
    data[92 * steps + cycle - 1] = row.word & 0xff;       // a->data[0]
    data[93 * steps + cycle - 1] = row.word >> 8 & 0xff;  // a->data[1]
    data[94 * steps + cycle - 1] = row.word >> 16 & 0xff; // a->data[2]
    data[95 * steps + cycle - 1] = row.word >> 24 & 0xff; // a->data[3]
    data[97 * steps + cycle - 1] = row.dirty;             // prevVerifier->dirty
  }
}

__global__ void
par_step_verify_mem(void* ctx, uint32_t steps, uint32_t count, Fp* arg0, Fp* arg1, Fp* arg2) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }

  if (cycle == 0 || is_par_safe_verify_mem(ctx, cycle)) {
    // printf("step_verify_mem(%u)\n", cycle);
    step_verify_mem(ctx, steps, cycle++, arg0, arg1, arg2, nullptr, nullptr);
    while (cycle < count && !is_par_safe_verify_mem(ctx, cycle)) {
      // printf("step_verify_mem(%u)\n", cycle);
      step_verify_mem(ctx, steps, cycle++, arg0, arg1, arg2, nullptr, nullptr);
    }
  }
}

__global__ void inject_backs_bytes(void* ctx, size_t steps, size_t count, Fp* data) {
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle == 0 || cycle >= count) {
    return;
  }

  MachineContext* mctx = static_cast<MachineContext*>(ctx);
  uint32_t writeCount = mctx->byteWrites[cycle - 1];
  // printf("inject> cycle: %u, writeCount: %u\n", cycle, writeCount);
  if (writeCount) {
    uint32_t pair = mctx->byteSorted[(cycle - 1) * kMaxBytePairsPerCycle + writeCount - 1];
    // printf("inject> pair: %x\n", pair);
    data[0 * steps + cycle - 1] = pair >> 8 & 0xff;
    data[1 * steps + cycle - 1] = pair & 0xff;
  }
}

extern "C" const char* risc0_circuit_rv32im_cuda_witgen(
    PreflightTrace* trace, uint32_t steps, uint32_t last_cycle, Fp* ctrl, Fp* io, Fp* data) {
  try {
    nvtx3::scoped_range range("witgen");

    // printf("risc0_circuit_rv32im_cuda_witgen\n");
    CUDA_OK(hipDeviceSynchronize());

    HostContext ctx(trace, steps);

    CudaStream stream;
    LaunchConfig cfg = getSimpleConfig(last_cycle);

    {
      // printf("step_exec\n");
      nvtx3::scoped_range range("step_exec");
      par_step_exec<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, ctrl, io, data);
      CUDA_OK(hipStreamSynchronize(stream));
    }

    ctx.ctx->sortRam();

    {
      // printf("inject_backs_ram\n");
      nvtx3::scoped_range range("inject_backs_ram");
      inject_backs_ram<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, data);
      CUDA_OK(hipStreamSynchronize(stream));
    }

    {
      // printf("step_verify_mem\n");
      nvtx3::scoped_range range("step_verify_mem");
      par_step_verify_mem<<<cfg.grid, cfg.block, 0, stream>>>(
          ctx.ctx, steps, last_cycle, ctrl, io, data);
      CUDA_OK(hipStreamSynchronize(stream));
    }

    ctx.ctx->sortBytes();

    {
      // printf("inject_backs_bytes\n");
      nvtx3::scoped_range range("inject_backs_bytes");
      inject_backs_bytes<<<cfg.grid, cfg.block, 0, stream>>>(ctx.ctx, steps, last_cycle, data);
      CUDA_OK(hipStreamSynchronize(stream));
    }

    {
      // printf("step_verify_bytes\n");
      nvtx3::scoped_range range("step_verify_bytes");
      step_verify_bytes<<<cfg.grid, cfg.block, 0, stream>>>(
          ctx.ctx, steps, last_cycle, ctrl, io, data, nullptr, nullptr);
      CUDA_OK(hipStreamSynchronize(stream));
    }

  } catch (const std::exception& err) {
    return strdup(err.what());
  }
  return nullptr;
}

__device__ void extern_halt(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  // no-op
}

__device__ void extern_trace(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  // no-op
}

__device__ void extern_getMajor(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  uint32_t major = trace->cycles[cycle].major;
  // printf("[%lu] getMajor: %u\n", cycle, major);
  outs[0] = major;
}

__device__ void extern_getMinor(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  // printf("getMinor\n");
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  outs[0] = trace->cycles[cycle].minor;
}

__device__ void extern_pageInfo(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  if (trace->isTrace) {
    printf("pageInfo\n");
  }
  size_t idx = trace->cycles[cycle].extraIdx;
  outs[0] = trace->extras[idx + 0];
  outs[1] = trace->extras[idx + 1];
  outs[2] = trace->extras[idx + 2];
}

__device__ void extern_ramWrite(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  // no-op
}

__device__ void extern_ramRead(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  uint32_t addr = args[0].asUInt32();
  size_t memIdx = trace->cycles[cycle].memIdx++;
  const MemoryTransaction& txn = trace->txns[memIdx];
  if (trace->isTrace) {
    printf("ramRead(%lu, 0x%x): txn(%u, 0x%x), memIdx: %lu\n",
           cycle,
           addr,
           txn.cycle,
           txn.addr,
           memIdx);
  }
  assert(cycle == txn.cycle && "Mismatched memory txn cycle");
  assert(addr == txn.addr && "Mismatched memory txn addr");
  outs[0] = txn.data & 0xff;
  outs[1] = txn.data >> 8 & 0xff;
  outs[2] = txn.data >> 16 & 0xff;
  outs[3] = txn.data >> 24 & 0xff;
}

__device__ void extern_syscallBody(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  size_t extraIdx = trace->cycles[cycle].extraIdx++;
  uint32_t word = trace->extras[extraIdx];
  outs[0] = word & 0xff;
  outs[1] = word >> 8 & 0xff;
  outs[2] = word >> 16 & 0xff;
  outs[3] = word >> 24 & 0xff;
}

__device__ void extern_syscallFini(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  PreflightTrace* trace = static_cast<MachineContext*>(ctx)->trace;
  size_t extraIdx = trace->cycles[cycle].extraIdx++;
  uint32_t a0 = trace->extras[extraIdx + 0];
  uint32_t a1 = trace->extras[extraIdx + 1];
  outs[0] = a0 & 0xff;
  outs[1] = a0 >> 8 & 0xff;
  outs[2] = a0 >> 16 & 0xff;
  outs[3] = a0 >> 24 & 0xff;
  outs[4] = a1 & 0xff;
  outs[5] = a1 >> 8 & 0xff;
  outs[6] = a1 >> 16 & 0xff;
  outs[7] = a1 >> 24 & 0xff;
}

__device__ void extern_divide(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  uint32_t numer = args[0].asUInt32() |       //
                   args[1].asUInt32() << 8 |  //
                   args[2].asUInt32() << 16 | //
                   args[3].asUInt32() << 24;
  uint32_t denom = args[4].asUInt32() |       //
                   args[5].asUInt32() << 8 |  //
                   args[6].asUInt32() << 16 | //
                   args[7].asUInt32() << 24;
  uint32_t sign = args[8].asUInt32();
  uint32_t onesComp = (sign == 2);
  bool negNumer = sign && int32_t(numer) < 0;
  bool negDenom = sign == 1 && int32_t(denom) < 0;
  if (negNumer) {
    numer = -numer - onesComp;
  }
  if (negDenom) {
    denom = -denom - onesComp;
  }
  uint32_t quot;
  uint32_t rem;
  if (denom == 0) {
    quot = 0xffffffff;
    rem = numer;
  } else {
    quot = numer / denom;
    rem = numer % denom;
  }
  uint32_t quotNegOut = (negNumer ^ negDenom) - ((denom == 0) * negNumer);
  uint32_t remNegOut = negNumer;
  if (quotNegOut) {
    quot = -quot - onesComp;
  }
  if (remNegOut) {
    rem = -rem - onesComp;
  }
  outs[0] = quot & 0xff;
  outs[1] = quot >> 8 & 0xff;
  outs[2] = quot >> 16 & 0xff;
  outs[3] = quot >> 24 & 0xff;
  outs[4] = rem & 0xff;
  outs[5] = rem >> 8 & 0xff;
  outs[6] = rem >> 16 & 0xff;
  outs[7] = rem >> 24 & 0xff;
}

__device__ void
extern_bigintQuotient(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  printf("bigintQuotient\n");
}

__device__ void
extern_plonkWrite_ram(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  MachineContext* mctx = static_cast<MachineContext*>(ctx);
  uint32_t addr = args[0].asUInt32();
  uint32_t memCycle = args[1].asUInt32();
  uint32_t memOp = args[2].asUInt32();
  uint32_t word = args[3].asUInt32() |       //
                  args[4].asUInt32() << 8 |  //
                  args[5].asUInt32() << 16 | //
                  args[6].asUInt32() << 24;
  if (mctx->trace->isTrace) {
    printf("plonkWriteRam(0x%x, %u, %u, 0x%x)\n", addr, memCycle, memOp, word);
  }
  uint32_t idx = mctx->ramIndex[cycle]++;
  assert(idx < kMaxRamRowsPerCycle);
  RamArgumentRow& row = mctx->ramRows[cycle * kMaxRamRowsPerCycle + idx];
  row.addr = addr;
  row.setCyclop(memCycle, memOp);
  row.word = word;
  row.dirty = 0;
}

void MachineContext::sortRam() {
  // printf("sortRam\n");
  nvtx3::scoped_range range("sortRam");
  std::vector<RamArgumentRow> compact;
  {
    nvtx3::scoped_range range("prepare");
    for (size_t cycle = 0; cycle < steps; cycle++) {
      size_t count = ramIndex[cycle];
      for (size_t i = 0; i < count; i++) {
        compact.push_back(ramRows[cycle * kMaxRamRowsPerCycle + i]);
      }
    }

    CUDA_OK(hipMallocManaged(&ramSorted, compact.size() * sizeof(RamArgumentRow)));
    CUDA_OK(hipMemcpy(ramSorted,
                       compact.data(),
                       compact.size() * sizeof(RamArgumentRow),
                       hipMemcpyHostToDevice));
  }

  {
    nvtx3::scoped_range range("sort");
    thrust::sort(thrust::device, ramSorted, ramSorted + compact.size());
  }

  {
    nvtx3::scoped_range range("dirty");
    uint32_t prevDirty = 0;
    for (size_t i = 0; i < compact.size(); i++) {
      RamArgumentRow& row = ramSorted[i];
      switch (row.getMemOp()) {
      case 0: // pageIo
        row.dirty = 0;
        break;
      case 1: // read
        row.dirty = prevDirty;
        break;
      case 2: // write
        row.dirty = 1;
        break;
      }
      prevDirty = row.dirty;
    }
  }

  {
    nvtx3::scoped_range range("update");
    thrust::exclusive_scan(thrust::device, ramIndex, ramIndex + steps, ramIndex, 0);
  }
}

__device__ void
extern_plonkRead_ram(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  MachineContext* mctx = static_cast<MachineContext*>(ctx);
  uint32_t idx = mctx->ramIndex[cycle]++;
  const RamArgumentRow& row = mctx->ramSorted[idx];
  outs[0] = row.addr;
  outs[1] = row.getMemCycle();
  outs[2] = row.getMemOp();
  outs[3] = row.word & 0xff;
  outs[4] = row.word >> 8 & 0xff;
  outs[5] = row.word >> 16 & 0xff;
  outs[6] = row.word >> 24 & 0xff;
}

__device__ void
extern_plonkWrite_bytes(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  // printf("plonkWrite_bytes\n");
  MachineContext* mctx = static_cast<MachineContext*>(ctx);
  uint32_t pair = args[0].asUInt32() << 8 | args[1].asUInt32();
  mctx->bytePairs[pair]++;
  mctx->byteWrites[cycle]++;
}

void MachineContext::sortBytes() {
  nvtx3::scoped_range range("sortBytes");
  // printf("sortBytes\n");

  size_t pos = 0;
  auto next = [&]() {
    while (!bytePairs[pos]) {
      pos++;
    }
    bytePairs[pos]--;
    return pos;
  };

  for (size_t cycle = 0; cycle < steps; cycle++) {
    uint32_t count = byteWrites[cycle];
    assert(count <= kMaxBytePairsPerCycle);
    for (size_t i = 0; i < count; i++) {
      byteSorted[cycle * kMaxBytePairsPerCycle + i] = next();
    }
  }
}

__device__ void
extern_plonkRead_bytes(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  MachineContext* mctx = static_cast<MachineContext*>(ctx);
  uint32_t idx = mctx->byteReads[cycle]++;
  uint32_t pair = mctx->byteSorted[cycle * kMaxBytePairsPerCycle + idx];
  // printf("plonkReadBytes> cycle: %lu, idx: %u, pair: %x\n", cycle, idx, pair);
  outs[0] = pair >> 8 & 0xff;
  outs[1] = pair & 0xff;
}

__device__ void extern_syscallInit(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  // no-op
}

__device__ void
extern_plonkWriteAccum_ram(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  AccumContext* actx = static_cast<AccumContext*>(ctx);
  actx->ram[cycle] = FpExt(args[0], args[1], args[2], args[3]);
}

__device__ void
extern_plonkWriteAccum_bytes(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  AccumContext* actx = static_cast<AccumContext*>(ctx);
  actx->bytes[cycle] = FpExt(args[0], args[1], args[2], args[3]);
}

__device__ void
extern_plonkReadAccum_ram(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  AccumContext* actx = static_cast<AccumContext*>(ctx);
  const FpExt& x = actx->ram[cycle];
  for (size_t i = 0; i < 4; i++) {
    outs[i] = x.elems[i];
  }
}

__device__ void
extern_plonkReadAccum_bytes(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  AccumContext* actx = static_cast<AccumContext*>(ctx);
  const FpExt& x = actx->bytes[cycle];
  for (size_t i = 0; i < 4; i++) {
    outs[i] = x.elems[i];
  }
}

__device__ void extern_log(void* ctx, size_t cycle, const char* extra, Fp* args, Fp* outs) {
  // printf("%s\n", extra);
}
